#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------
   Author: Thomas Nowotny
  
   Institute: Informatics
              University of Sussex 
              Brighton BN1 9QJ, UK
  
   email to:  t.nowotny@sussex.ac.uk
  
   initial version: 2014-06-26
  
--------------------------------------------------------------------------*/

//--------------------------------------------------------------------------
/*! \file VClampGA.cu

\brief Main entry point for the GeNN project demonstrating realtime fitting of a neuron with a GA running mostly on the GPU. 
*/
//--------------------------------------------------------------------------

#include "VClampGA.h"

//--------------------------------------------------------------------------
/*! \brief This function is the entry point for running the project
*/
//--------------------------------------------------------------------------

int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    fprintf(stderr, "usage: VClampGA <basename> <CPU=0, GPU=1> \n");
    return 1;
  }
  int which= atoi(argv[2]);
  string OutDir = toString(argv[1]) +"_output";
  string name;
  name= OutDir+ "/"+ toString(argv[1]) + toString(".time");
  FILE *timef= fopen(name.c_str(),"a");  

  timer.startTimer();
  write_para();
  
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.I"); 
  FILE *osf= fopen(name.c_str(),"w");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.err"); 
  FILE *ose= fopen(name.c_str(),"w");
  name= OutDir+ "/"+ toString(argv[1]) + toString(".out.best"); 
  FILE *osb= fopen(name.c_str(),"w");

  //-----------------------------------------------------------------
  // build the neuronal circuitery

  NNmodel model;
  modelDefinition(model);
  allocateMem();
  initialize();
  var_reinit(1.0);         // this includes copying vars for the GPU version
  initexpHH();
  fprintf(stderr, "# neuronal circuitery built, start computation ... \n\n");

  //------------------------------------------------------------------
  // output general parameters to output file and start the simulation

  fprintf(stderr, "# We are running with fixed time step %f \n", DT);

  int done= 0, sn;
  unsigned int VSize= NPOP*sizeof(double);
  double lt, oldt;
  inputSpec I;
  initI(I);
  stepVGHH= I.baseV;
  int iTN= (int) (I.t/DT);

  t= 0.0;
  while (!done) 
  {
    truevar_init();
    truevar_initexpHH();
    lt= 0.0;
    sn= 0;	
    for (int iT= 0; iT < iTN; iT++) {
      oldt= lt;
      runexpHH(t); 
      if (which == GPU) {
	stepTimeGPU(t);
      }
      else {
	stepTimeCPU(t);
      }
      t+= DT;	
      lt+= DT;
      // CHECK_CUDA_ERRORS(hipMemcpy(VHH, d_VHH, VSize, hipMemcpyDeviceToHost));
      // fprintf(osf,"%f %f %f ", t, stepVGHH, IsynGHH);
      // for (int i= 0; i < NPOP; i++) {
      //   fprintf(osf, "%f ", 1000.0*(stepVGHH-VHH[i]));
      // }
      // fprintf(osf, "\n");
      if ((sn < I.N) && (oldt < I.st[sn]) && (lt >= I.st[sn])) {
	stepVGHH= I.V[sn];
	sn++;
      }
    }
    CHECK_CUDA_ERRORS(hipMemcpy(errHH, d_errHH, VSize, hipMemcpyDeviceToHost));
    fprintf(ose,"%f ", t);
    for (int i= 0; i < NPOP; i++) {
       fprintf(ose, "%f ", errHH[i]);
    }
    fprintf(ose,"\n");
    fprintf(osb, "%f %f %f %f %f %f %f ", gNaexp, ENaexp, gKexp, EKexp, glexp, Elexp, Cexp);
    procreatePop(osb);
    gNaexp= myHH_ini[4]+40*sin(3.1415927*t/40000);
//    ENaexp= myHH_ini[5]+40*sin(3.1415927*t/40000);
    cerr << "% " << t << endl;
    done= (t >= TOTALT);
  }
  timer.stopTimer();
  fprintf(timef,"%f \n",timer.getElapsedTime());  
  // close files 
  fclose(osf);
  fclose(ose);
  fclose(timef);
  fclose(osb);
  freeDeviceMem();
  hipDeviceReset();
  return 0;
}
